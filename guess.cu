#include "hip/hip_runtime.h"
#include "check_sol.h"
#include "check_thread.h"
#include "guess.h"
#include "partial_eval.h"
#include "read_sys.h"

#include "cuda_util.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

//
#define NUM_THREADS (1 << (N - K))
#define BLOCK_DIM (NUM_THREADS > 128 ? 128 : NUM_THREADS)
#define GRID_DIM (NUM_THREADS / BLOCK_DIM)

//
#define KERNEL_SOLUTIONS (1u << 2u)
#define KERNEL_SHARED_SOLUTIONS (1u << 2u)


#define PRINT_SOL(X) printf("%lX\n", X)
// #define PRINT_SOL(X)

#define LOG(level, f_, ...) fprintf(stdout, (f_), ##__VA_ARGS__)
// #define LOG(level, f_, ...)

extern "C" double get_ms_time(void) {
	struct timeval timev;

	gettimeofday(&timev, NULL);
	return (double) timev.tv_sec * 1000 + (double) timev.tv_usec / 1000;
}

__device__ __constant__ uint32_t deg2_block[MAX_K * (MAX_K - 1) / 2];

//template<const uint32_t tpb>
//__global__ void guess(const uint32_t *deg1, uint32_t *result,
//		const uint32_t num_threads, const uint32_t k);

#include "kernel_opt_shared.cuh"


__device__
uint32_t eval(const uint32_t *sys, const uint64_t sol, const uint32_t N, const uint32_t M) {
  uint32_t i, j, pos = 0;
  uint32_t x[64], check = 0;

  for (uint32_t b = 0; b < M; b += 32) {
    uint32_t mask = (M - b) >= 32 ? 0xffffffff : ((1 << (M - b)) - 1);

    for (i = 0; i < N; i++)
      x[i] = ((sol >> i) & 1) ? mask : 0;

    // computing quadratic part
    for (j = 1; j < N; j++)
      for (i = 0; i < j; i++)
        check ^= sys[pos++] & x[i] & x[j];

    // computing linear part
    for (i = 0; i < N; i++)
      check ^= sys[pos++] & x[i];

    // constant part
    check ^= sys[pos++];
  }

  return check;
}

__global__
void check(uint64_t *sol_out, const uint32_t *sys, const uint32_t *result,
		   const uint32_t N, const uint32_t M, const uint32_t K) {
	const uint32_t tid = (blockIdx.x*blockDim.x + threadIdx.x);

	for (uint32_t i = 0; i < KERNEL_SOLUTIONS; i++) {
		const uint64_t x = uint64_t(tid << K) | result[tid*KERNEL_SOLUTIONS + i];
		if (eval(sys, x, N, M) == 0) {
			// printf("found %d %lu\n", tid, x);
			*sol_out = x;
			//atomicExch(sol_out, x);
			break;
		}
	}
}

static int cuda_device = 0;
static bool init = false;

extern "C" void setDevice(int device) {
	cuda_device = device;
	init = false;
}

extern "C"
uint64_t searchSolution(uint32_t *coefficients, 
						unsigned int number_of_variables,
						unsigned int number_of_equations) {

	if (!init) {
		double initTime = 0;
		initTime -= get_ms_time();

		// set to designated device
		// int test;
		CUDA_ASSERT(hipSetDevice(cuda_device));
		// hipGetDevice(&test);
		// assert(atoi(argv[1]) == test);

		initTime += get_ms_time();
		//LOG(INFO, "init time = %f\n", initTime);

		init = true;
	}

	double preTime = 0, memTime = 0, recvTime = 0, checkTime = 0, ctTime = 0;
	float kernelTime = 0;
	uint32_t solCount = 0, ctCount = 0;

	uint64_t res = UINT64_MAX;

	// create events here
	hipEvent_t start, stop;
	CUDA_ASSERT(hipEventCreate(&start));
	CUDA_ASSERT(hipEventCreate(&stop));
	CUDA_ASSERT(hipDeviceSynchronize());

	uint32_t N = number_of_variables;
	uint32_t M = number_of_equations;

	uint32_t K = 32;

	if (K > MAX_K)
		K = MAX_K;

	if (N <= K)
		K = N - 1;
	
	const uint32_t sizeofsys = (1 + N + N*(N-1)) * (1+(M > 32)) * sizeof(uint32_t);
	uint32_t *sys = pack_sys_data(coefficients, N, M), *dsys;
	CUDA_ASSERT(hipMalloc(&dsys, sizeofsys));
	CUDA_ASSERT(hipMemcpy(dsys, sys, sizeofsys, hipMemcpyHostToDevice));

	preTime -= get_ms_time(); // partial evaluation

	cudaData<uint32_t> deg1((K + 1) * NUM_THREADS);

	partial_eval(sys, deg1.host, N, K);

	preTime += get_ms_time();

	memTime -= get_ms_time(); // initializing GPU memory space

	// initialize constant memory space for the quadratic part
	CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(deg2_block), sys, sizeof(uint32_t) * K * (K - 1) / 2));
	CUDA_ASSERT(hipDeviceSynchronize());

	// initialize global memory space for the linear parts
	deg1.write();

	// initialize global memory space for the results of each threads
	cudaData<uint32_t> result(NUM_THREADS * KERNEL_SOLUTIONS);

	memTime += get_ms_time();

	// launch kernel function and measure the elapsed time
	hipEventRecord(start, 0);

	// guess<<<GRID_DIM, BLOCK_DIM>>>(deg1.dev, result.dev, NUM_THREADS, K);
	guess<<<GRID_DIM, BLOCK_DIM>>>((const uint32_t *)deg1.dev, result.dev, uint32_t(NUM_THREADS), K);
	CUDA_ASSERT(hipDeviceSynchronize());

	CUDA_ASSERT(hipEventRecord(stop, 0));
	CUDA_ASSERT(hipEventSynchronize(stop));

	CUDA_ASSERT(hipEventElapsedTime(&kernelTime, start, stop));
	CUDA_ASSERT(hipDeviceSynchronize());

	recvTime -= get_ms_time(); // copy the results of each thread to host

#if 1
	result.read();
#endif

	recvTime += get_ms_time();

	checkTime -= get_ms_time(); // check if the results are available

	uint64_t ans;

#if 0
	uint64_t *dans;
	CUDA_ASSERT(hipMalloc(&dans, 8));
	check<<<GRID_DIM, BLOCK_DIM>>>
		(dans, dsys, result.dev, N, M, K);
	CUDA_ASSERT(hipDeviceSynchronize());

	CUDA_ASSERT(hipMemcpy(&ans, dans, 8, hipMemcpyDeviceToHost));
	// 	assert(check_sol(sys, ans, N, M) == 0);
#else

	for (uint64_t i = 0; i < NUM_THREADS; i++) {
		for (uint32_t j = 0; j < KERNEL_SOLUTIONS; j++) {
			ans = result.host[i*KERNEL_SOLUTIONS + j];
			if (check_sol(sys, (i << K) | ans, N, M) == 0) {
				solCount++;
				LOG(INFO, "thread %lX ---------> one solution %X\n", i, ans);
				PRINT_SOL((i << K) | ans);

				res = (i << K) | ans;

				goto end;
			}
		}
	}
end:
#endif
	checkTime += get_ms_time();

	float totalTime = preTime + memTime + kernelTime + recvTime + checkTime;

	// print the time for each step
	//LOG(INFO, "partial ");
	//LOG(INFO, "mem ");
	//LOG(INFO, "kernel ");
	//LOG(INFO, "recv ");
	//LOG(INFO, "check #sol ");
	//LOG(INFO, "(mult sol: t #ct)\n");
	LOG(INFO, "%.3f ", preTime);
	LOG(INFO, "%.3f ", memTime);
	LOG(INFO, "%.3f ", kernelTime);
	LOG(INFO, "%.3f ", recvTime);
	LOG(INFO, "%.3f ", checkTime);
	LOG(INFO, "%u ", solCount);
	LOG(INFO, "(%.3f  %u) ", ctTime, ctCount);

	LOG(INFO, "%.3f \n", totalTime);
	// release memory spaces
	free(sys);

	return res;
}
