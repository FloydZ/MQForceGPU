#include "hip/hip_runtime.h"
#include "check_sol.h"
#include "check_thread.h"
#include "guess.h"
#include "partial_eval.h"
#include "read_sys.h"

#include "cuda_util.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

//
#define NUM_THREADS (1 << (N - K))
#define BLOCK_DIM (NUM_THREADS > 128 ? 128 : NUM_THREADS)
#define GRID_DIM (NUM_THREADS / BLOCK_DIM)

//
#define KERNEL_SOLUTIONS (1u << 2u)
#define KERNEL_SHARED_SOLUTIONS (1u << 2u)


#define PRINT_SOL(X) printf("%lX\n", X)
// #define PRINT_SOL(X)

#define LOG(level, f_, ...) fprintf(stdout, (f_), ##__VA_ARGS__)
// #define LOG(level, f_, ...)

extern "C" double get_ms_time(void) {
	struct timeval timev;

	gettimeofday(&timev, NULL);
	return (double) timev.tv_sec * 1000 + (double) timev.tv_usec / 1000;
}

__device__ __constant__ uint32_t deg2_block[MAX_K * (MAX_K - 1) / 2];

//template<const uint32_t tpb>
//__global__ void guess(const uint32_t *deg1, uint32_t *result,
//		const uint32_t num_threads, const uint32_t k);

#include "kernel_opt_shared.cuh"

static int cuda_device = 0;
static bool init = false;

extern "C" void setDevice(int device) {
	cuda_device = device;
	init = false;
}

extern "C"
uint64_t searchSolution(uint32_t *coefficients, unsigned int number_of_variables,
		unsigned int number_of_equations) {

	if (!init) {
		double initTime = 0;
		initTime -= get_ms_time();

		// set to designated device
		// int test;
		CUDA_ASSERT(hipSetDevice(cuda_device));
		// hipGetDevice(&test);
		// assert(atoi(argv[1]) == test);

		initTime += get_ms_time();
		//LOG(INFO, "init time = %f\n", initTime);

		init = true;
	}

	double preTime = 0, memTime = 0, recvTime = 0, checkTime = 0, ctTime = 0;
	float kernelTime = 0;
	uint32_t solCount = 0, ctCount = 0;

	uint64_t res = UINT64_MAX;

	// create events here
	hipEvent_t start, stop;
	CUDA_ASSERT(hipEventCreate(&start));
	CUDA_ASSERT(hipEventCreate(&stop));
	CUDA_ASSERT(hipDeviceSynchronize());

	uint32_t N = number_of_variables;
	uint32_t M = number_of_equations;

	uint32_t K = 32;

	if (K > MAX_K)
		K = MAX_K;

	if (N <= K)
		K = N - 1;

	uint32_t *sys = pack_sys_data(coefficients, N, M);

	preTime -= get_ms_time(); // partial evaluation

	cudaData<uint32_t> deg1((K + 1) * NUM_THREADS);

	partial_eval(sys, deg1.host, N, K);

	preTime += get_ms_time();

	memTime -= get_ms_time(); // initializing GPU memory space

	// initialize constant memory space for the quadratic part
	CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(deg2_block), sys, sizeof(uint32_t) * K * (K - 1) / 2));
	CUDA_ASSERT(hipDeviceSynchronize());

	// initialize global memory space for the linear parts
	deg1.write();

	// initialize global memory space for the results of each threads
	cudaData<uint32_t> result(NUM_THREADS * KERNEL_SOLUTIONS);

	memTime += get_ms_time();

	// launch kernel function and measure the elapsed time
	hipEventRecord(start, 0);

	// guess<<<GRID_DIM, BLOCK_DIM>>>(deg1.dev, result.dev, NUM_THREADS, K);
	guess<<<GRID_DIM, BLOCK_DIM>>>((const uint32_t *)deg1.dev, result.dev, uint32_t(NUM_THREADS), K);
	CUDA_ASSERT(hipDeviceSynchronize());

	CUDA_ASSERT(hipEventRecord(stop, 0));
	CUDA_ASSERT(hipEventSynchronize(stop));

	CUDA_ASSERT(hipEventElapsedTime(&kernelTime, start, stop));
	CUDA_ASSERT(hipDeviceSynchronize());

	recvTime -= get_ms_time(); // copy the results of each thread to host

	result.read();

	recvTime += get_ms_time();

	checkTime -= get_ms_time(); // check if the results are available

	int32_t ans;

	for (uint64_t i = 0; i < NUM_THREADS; i++) {
		for (uint32_t j = 0; j < KERNEL_SOLUTIONS; j++) {
			ans = result.host[i*KERNEL_SOLUTIONS + j];
			if (check_sol(sys, (i << k) | ans, N, M) == 0) {
				solCount++;
				// LOG(INFO, "thread %lX ---------> one solution %X\n", i, ans);
				// PRINT_SOL((i << K) | ans);

				res = (i << K) | ans;

				goto end;
			}
		}

		if (deg1.host[K * NUM_THREADS + i] == 0) // special case: check for (prtial) zero solution
		{
			solCount++;
			if (check_sol(sys, (i << K) | 0, N, M) == 0) {
				// LOG(INFO, "thread %lX ---------> one solution 0\n", i);
				// PRINT_SOL(i << K);

				res = (i << K);

				goto end;
			}
		}
	}

end:

	checkTime += get_ms_time();

	float totalTime = preTime + memTime + kernelTime + recvTime + checkTime;

	// print the time for each step
	//LOG(INFO, "partial ");
	//LOG(INFO, "mem ");
	//LOG(INFO, "kernel ");
	//LOG(INFO, "recv ");
	//LOG(INFO, "check #sol ");
	//LOG(INFO, "(mult sol: t #ct)\n");
	LOG(INFO, "%.3f ", preTime);
	LOG(INFO, "%.3f ", memTime);
	LOG(INFO, "%.3f ", kernelTime);
	LOG(INFO, "%.3f ", recvTime);
	LOG(INFO, "%.3f ", checkTime);
	LOG(INFO, "%u ", solCount);
	LOG(INFO, "(%.3f  %u) ", ctTime, ctCount);

	LOG(INFO, "%.3f \n", totalTime);
	// release memory spaces
	free(sys);

	return res;
}
